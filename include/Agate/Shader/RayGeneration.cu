#include "hip/hip_runtime.h"
﻿#include <optix.h>

#include "LaunchParameter.h"

namespace Agate {

extern "C" __constant__ OptixLaunchParams optixLaunchParams;

extern "C" __global__ void __raygen__renderFrame()
{
    const int frameID = optixLaunchParams.frameID;

    const uint3 theLaunchIndex = optixGetLaunchIndex();
    if (frameID == 0 &&
        theLaunchIndex.x == 0 &&
        theLaunchIndex.y == 0) {
        // we could of course also have used optixGetLaunchDims to query
        // the launch size, but accessing the optixLaunchParams here
        // makes sure they're not getting optimized away (because
        // otherwise they'd not get used)
        printf("############################################\n");
        printf("Hello world from OptiX 7 raygen program!\n(within a %ix%i-sized launch)\n",
               optixLaunchParams.frame_buffer_size.x,
               optixLaunchParams.frame_buffer_size.y);
        printf("############################################\n");
    }

    // ------------------------------------------------------------------
    // for this example, produce a simple test pattern:
    // ------------------------------------------------------------------

    // compute a test pattern based on pixel ID
    const int ix = theLaunchIndex.x;
    const int iy = theLaunchIndex.y;

    const int r = ((ix + frameID) % 256);
    const int g = ((iy + frameID) % 256);
    const int b = ((ix + iy + frameID) % 256);

    // convert to 32-bit rgba value (we explicitly set alpha to 0xff
    // to make stb_image_write happy ...
    const uint32_t rgba = 0xff000000
        | (r << 0) | (g << 8) | (b << 16);

    // and write to frame buffer ...
    const uint32_t fbIndex = ix + iy * optixLaunchParams.frame_buffer_size.x;
    optixLaunchParams.color_buffer[fbIndex] = rgba;
}

} // namespace Agate
